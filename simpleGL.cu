#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/
#include <typeinfo>


//#include "main.h"
//#include "mesh.h"
//#include "data.h"
//#include "simpleGL.h"


// includes, system
//#include "simpleGL.h"
// try to add a global struct for reference to internal routines
/*
PPart *host_P;
PPart *dev_P;
MMesh *dev_MM;
MMesh *MM;
DData *dev_DD;
DData *DD;
int DD3[4];
*/

//void move(float4 *pos, struct PPart *PP,struct MMesh *MM, struct DData *DD, float time_now);

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////




void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL()
{
    int argc = 1;
    char *argv[1] = {(char*)"Something"};
    glutInit(&argc, argv);
    //glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Regular NetCDF Particle Tracking");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0050, 0.005, 0.0050, 1.0);  // background color  0,0,0,1 is black
    glColor4f(0.0,1.0,0.0,1.0);   // set color
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}


////////////////////////////////////////////////////////////////////////////////
//  Initialize a few gl, timer and cuda 
//  then start gl looping to call function display
////////////////////////////////////////////////////////////////////////////////
bool GLmoveparticle(struct PPart *PP, struct MMesh *MM, struct DData *DD)
{
    //int DD3[4];
    // Create the CUTIL timer
    sdkCreateTimer(&timer);
    printf("TFG GLmoveparticle   starting...\n");
    printf("TFG GLmoveparticle DD[0].time      %g %g %g %g\n",DD[0].time/3600,DD[1].time/3600,DD[2].time/3600,DD[3].time/3600);
    printf("TFG GLmoveparticle DD[0].DD3      %d %d %d %d\n",DD[0].DD3[0],DD[0].DD3[1],DD[0].DD3[2],DD[0].DD3[3]);
    g_time_now = (DD[0].time + DD[1].time)/2.0; 

    //initial the cudaDevice to use, as if there is a choice?
    hipDeviceProp_t deviceProp;
    int devID = gpuGetMaxGflopsDeviceId();
    checkCudaErrors(hipSetDevice(devID));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n"
         , devID, deviceProp.name, deviceProp.major, deviceProp.minor);


        // First initialize OpenGL context, so we can properly set the GL for CUDA.
        // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
        //if (false == initGL(&argc, argv))        
        if (false == initGL())

        {
            return false;
        }

        // register callbacks.   these are locally defined functions
        glutDisplayFunc(display);
        glutKeyboardFunc(keyboard);
        glutMouseFunc(mouse);
        glutMotionFunc(motion);
#if defined (__APPLE__) || defined(MACOSX)
        atexit(cleanup);
#else
        glutCloseFunc(cleanup);
#endif

        // create VBO
        createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

        // size_t DDSizeGeneral = sizeof(DData)*4;
        // hipMemcpy(DD, dev_DD,DDSizeGeneral,hipMemcpyDeviceToHost);


        // run the cuda part from routine display 
        // specified in glutDisplayFunc(display);
        // which is triggered by glutMainLoop
        //runCuda(&cuda_vbo_resource);

        // start rendering mainloop
        printf(" Start glutMainLoop  >display>runCuda \n\n");

        glutMainLoop();

        printf(" Return from glutMainLoop\n");

//    }

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation,  called from display
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
    //printf("TFG runCuda host_P[10].x_present %g\n",host_P[10].x_present);
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    float time_now;

    size_t DDSizeGeneral = sizeof(DData)*4;
    size_t MMSizeGeneral = sizeof(MMesh)*4;

    if (iDD==-1){   // First update, need to localize DD, MM only once
                    // initialized in simpleGL.h, global to this file
                    
        printf("\n runCuda First Pass\n");
            try {
                printf(" Can I print DD[0].time_now %g\n",DD[0].time_now);
            } catch (const std::runtime_error& e){
                printf(" Error on print DD[0].time_now Message: %s\n",e.what());
            }
        hipMemcpy(DD, dev_DD,DDSizeGeneral,hipMemcpyDeviceToHost);
        hipMemcpy(MM, dev_MM,MMSizeGeneral,hipMemcpyDeviceToHost);
                printf(" After hipMemcpy  DD[0].time_now %fsec %f hr\n",DD[0].time_now,DD[0].time_now/3600.);

        iDD=0;
    }
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
    *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);
    

    //int DD33=1;   256,64
    move3d<<< 256,64 >>>(dptr,dev_P,dev_MM,dev_DD);
    hipDeviceSynchronize();
    DD[0].time_now += CUDA_STEPS* DT_SEC;   // 0.01f;   
    time_now = DD[0].time_now;
//printf("After cuda move3d time_now = %fs %ghr\n",time_now, time_now/3600.);
    float time_frac=(time_now - DD[DD[0].DD3[0]].time)/(DD[DD[0].DD3[2]].time - DD[DD[0].DD3[0]].time);
    bool timetest =  (time_frac > .75);
    if (timetest ){
        //  Every hour a new data file is needed. Read dev_DD to obtain time_now
        
        // Assume or test that the fourth ReadData thread is finished and move to dev_DD  BROKEN
        hipMemcpy(dev_DD,DD,DDSizeGeneral,hipMemcpyHostToDevice);

        //  Update DD3  
        for (int i=0; i<4 ; i++)DD[0].DD3[i]=(DD[0].DD3[i]+1)%4;

        // DD3[3] is next spot to be updated, will be updated in this section
        //  Thread this off to execute while elsewhere.
        //        printf(" DD[# 1].time = %g %g %g %g\n",DD[0].time/3600.,DD[1].time/3600.,DD[2].time/3600.,DD[3].time/3600.);
        
        //  New generated filename routine:
        DD[0].ToDay +=3600;  // for hourly files
        string newername = NetCDFfiledate(DD[0].filetemplate,DD);

        bool RunThreadRead = true;
        if (RunThreadRead){
            std::thread t1(ReadFieldNetCDF, std::ref(newername),std::ref(DD[0].DD3[3]),
            std::ref(DD),std::ref(MM) );
            t1.join();   // Wait here for thread to finish. Makes threading moot.  Testing only.
            //t1.detach();    // Let it loose, but with no test for finished crashes
            }
        else{
            ReadFieldNetCDF(newername,DD[0].DD3[3],DD,MM);
            }
/*
// List Nvidia resources to see if it is growing.
//checkGpuMem();
float free_m,total_m,used_m;
size_t free_t,total_t;
hipMemGetInfo(&free_t,&total_t);
free_m =(uint)free_t/1048576.0 ;
total_m=(uint)total_t/1048576.0;
used_m=total_m-free_m;
printf ( "  mem free %ld .... %f MB \n  mem total %ld....%f MB mem used %f MB\n"
  ,free_t,free_m,total_t,total_m,used_m);
*/
        float dhr=3600.;
        printf(" DD[     0:3].time = %g %g %g %g\n",DD[0].time/dhr,DD[1].time/dhr,DD[2].time/dhr,DD[3].time/dhr);
        printf(" DD[DDT[0:3]].time = %g %g %g %g\n",
           DD[DD[0].DD3[0]].time/dhr,DD[DD[0].DD3[1]].time/dhr,DD[DD[0].DD3[2]].time/dhr,DD[DD[0].DD3[3]].time/dhr);

        iDD+=1;
        printf(" iDD = %d\n\n",iDD);
    }    // End of hourly DD update

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}


////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    size = MAX_GLPARTICLES *4*sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(translate_x, translate_y, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(0.0, 1.0, 0.0);     // Color of points

    glPointSize(1);
    glEnable(GL_POINT_SMOOTH);
    glEnable(GL_BLEND);

    //glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glDrawArrays(GL_POINTS, 0, MAX_GLPARTICLES);

    glDisableClientState(GL_VERTEX_ARRAY);
    char buffer[25] ;
    float myFloat = g_time_now/3600.;   // convert to hours
    int ret =snprintf(buffer, sizeof buffer, "time_now = %.2f hr", myFloat);
    //and more";
    //char hello[] = str;
    //glutSetWindowTitle(buffer);
    
    /*if ( myFloat > 5.) {
        glutDestroyWindow(glutGetWindow());
        return;
        }
    */
    glutSwapBuffers();

    sdkStopTimer(&timer);
    computeFPS();    

}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else
                glutDestroyWindow(glutGetWindow());
                return;
            #endif
        case (112) : 
        { std::chrono::milliseconds timespan(5000); std::this_thread::sleep_for(timespan);}    // sleep for 5sec = 5000ms
        break ;
        case (104) : 
        {printf("\nesc = stop\n p = 5sec pause\n h = this help\n");} break;
    }
    printf("key = %d\n",key);  // p pause is 112
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)     // Rotate around x and y axis pitch and yaw
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 2) // magnification  z axis move push down on scroll button and move mouse
    {
        translate_z += dy * 0.01f;
    }
    else if(mouse_buttons & 4)    // Translate side to side or up and down
    { //printf("mouse button 2\n");
        translate_x += dx * 0.01f;
        translate_y -= dy * 0.01f;}

    else if(mouse_buttons & 3)
    { printf("mouse button 3\n");}
    else if(mouse_buttons & 0)
    { printf("mouse button 0\n");}
    //else 
    //   printf(" else mouse button = %d\n",mouse_buttons);

    mouse_old_x = x;
    mouse_old_y = y;
}



//Fancy cuda kernel can be called using dev_P, dev_MM, dev_DD   but define it with local names
// move<<<  >>> ( pos,dev_P,dev_MM,dev_DD);

__global__
void move(float4 *pos, struct PPart *PP,struct MMesh *MM, struct DData *DD){

//Make sure that PP will fit into pos
    //unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    //unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
unsigned int maxGLnum = blockDim.x*blockDim.y*gridDim.x*gridDim.y;
maxGLnum = MAX_GLPARTICLES;
float scale = SCALE_GL;
unsigned int Ipx;
int DDT0, DDT2;
int DD3[3];
DD3[0]=DD[0].DD3[0];
DD3[1]=DD[0].DD3[1];
DD3[2]=DD[0].DD3[2];
DDT0=DD3[0];
//DDT1=DD3[1];
DDT2=DD3[2];

/*   real stuff now  */
int DeBuG = false;   //   true or false
int DeBuGIP = NUM_PARTICLES/2;
int i_ele, keepgoing, k;
float xpart, ypart;
float smallest_value = -0.01000; // -0.001; 
//float time_now;
// float      now a passed argument
float time_now = DD[0].time_now;    // Will use dev_DD after the first pass with new DD
float time_frac=(time_now - DD[DDT0].time)/(DD[DDT2].time - DD[DDT0].time);
double dt_sec=DT_SEC;
int igrounded;

//  Cuda strides
int cudaindex = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;
 
if (cudaindex==-1 | cudaindex==-2){  //  0 2500
    printf(" t.x=%d blockIdx.x=%d blockDim.x=%d gridDim.x=%d cudaindex=%d stride=%d\n",
threadIdx.x,blockIdx.x,blockDim.x, gridDim.x,cudaindex,stride);
}
// Main time loop. Loop CUDA_STEPS times between returns for plotting
for (int itime=0; itime<CUDA_STEPS; itime++){

//loop on particles for this cuda up to max PP[0].num_P
    igrounded =0;
for(int Ip = cudaindex; Ip <NUM_PARTICLES; Ip += stride){

  // Find surrounding triangle of Particle
  i_ele = PP[Ip].i_ele;
  xpart = PP[Ip].x_present;
  ypart = PP[Ip].y_present;

  if (DeBuG && (itime%100==0) && Ip==DeBuGIP) 
  {printf(" Move itime=%d i_ele=%d, PP.i_ele=%d,  MM[0].ele0=%ld, MM[0].ele2=%ld MM[0].ele2=%ld\n",
  itime, i_ele,PP[Ip].i_ele,MM[0].ele[i_ele][0],MM[0].ele[i_ele][1],MM[0].ele[i_ele][2]);}
  

    //  Check for out of domain/ grounded particle
    //  do work if in-domain  else increment igrounded and skip main part of move
if (i_ele >= 0) { 

  keepgoing = 1; 
  while (keepgoing  == 1){


//  if any of the f's are negative, walk that way and restart while loop
   k=0;
   PP[Ip].factor[k]=MM[0].a_frac[i_ele][k]*xpart + 
      MM[0].b_frac[i_ele][k]*ypart + MM[0].c_frac[i_ele][k];
   if ( PP[Ip].factor[k] < smallest_value) { 
   	i_ele = MM[0].tri_connect[i_ele][0]; 
   }
   else { 
      k=1;
      PP[Ip].factor[k]=MM[0].a_frac[i_ele][k]*xpart + MM[0].b_frac[i_ele][k]*ypart + MM[0].c_frac[i_ele][k];
      if ( PP[Ip].factor[k] < smallest_value ) { 
      	  i_ele = MM[0].tri_connect[i_ele][1] ; 
      }
      else { 
          k=2;
          PP[Ip].factor[k]=MM[0].a_frac[i_ele][k]*xpart + MM[0].b_frac[i_ele][k]*ypart + MM[0].c_frac[i_ele][k];
          if ( PP[Ip].factor[k] < smallest_value ) { 
	  	i_ele = MM[0].tri_connect[i_ele][2] ;
          }
          else {
             //  Found it, iele,   all f's are positive 
             keepgoing = 0;
	  }
      }
   }
   if (i_ele < 0) {    // newly grounded particle, zero him out.
     	PP[Ip].factor[0]=0.0; PP[Ip].factor[1]=0.0; PP[Ip].factor[2]=0.0;
         keepgoing = 0;
         igrounded++;
         // newly grounded
         PP[Ip].i_ele = i_ele;
   }
 }   // end of while keepgoing 

if (i_ele>=0){     // good particle still in the mesh
 PP[Ip].i_ele =i_ele; 

 // moveing through particle.move
   float factor0=PP[Ip].factor[0];
   float factor1=PP[Ip].factor[1];
   float factor2=PP[Ip].factor[2];
 
   // i_ele is element, ele1 is node index of corner
   //  node = ele[ele_index][corner_index] 
   long ele0 = MM[0].ele[i_ele][0];
   long ele1 = MM[0].ele[i_ele][1];
   long ele2 = MM[0].ele[i_ele][2];

// Found new i_ele , ele0, ele1, ele2 and have factor1, factor2, factor0

  int sigma_level=0;   // Need to upgrade to 3d sometime....
  double u[3];   // velocities at corners of surrounding triangle
  double v[3];
  double w[3];
  float a,b,c;
// UVW space interpolation from the three surrounding points[ele 012]
  for (int i=0; i<3; i++){
    u[i]= factor0 * DD[DD3[i]].U[sigma_level][ele0] 
    + factor1*DD[DD3[i]].U[sigma_level][ele1]
    + factor2*DD[DD3[i]].U[sigma_level][ele2];    
    v[i]= factor0 * DD[DD3[i]].V[sigma_level][ele0] 
    + factor1*DD[DD3[i]].V[sigma_level][ele1]
    + factor2*DD[DD3[i]].V[sigma_level][ele2];
    w[i]= factor0 * DD[DD3[i]].W[sigma_level][ele0] 
    + factor1*DD[DD3[i]].W[sigma_level][ele1]
    + factor2*DD[DD3[i]].W[sigma_level][ele2];
    }
  // formula for quadratic time interpolation of three points, assuming equal spacing
  //  time_frac = 0  at DD[0].time,  time_frac = .5 DD[1].time,  time-frac = 1.0 DD[2].time
  a =  2.*u[2] -4.*u[1] +2.*u[0];
  b = -   u[2] +4.*u[1] +   u[0];
  c =                       u[0];
  float Upnow = a*time_frac*time_frac + b*time_frac + c;

  a =  2.*v[2] -4.*v[1] +2.*v[0];
  b = -   v[2] +4.*v[1] +   v[0];
  c =                       v[0];   
  float Vpnow = a*time_frac*time_frac + b*time_frac + c;

  a =  2.*w[2] -4.*w[1] +2.*w[0];
  b = -   w[2] +4.*w[1] +   w[0];
  c =                       w[0];   
  float Wpnow = a*time_frac*time_frac + b*time_frac + c; 


  /*  Now have time and space interpolates of U,V,W for particle */
  /* Apply them to the particle coordinates and done! 
   (unless temporal runge kutta is needed. 
    Running goofy small time steps)*/

    PP[Ip].x_present += dt_sec*(Upnow*1.) ; 
    PP[Ip].y_present += dt_sec*(Vpnow*1.); 
    PP[Ip].z_present += dt_sec*Wpnow*1.;
    
    // using an Xbox from Particles in meters
    /*float shrinkage = 1.0;   //dec 4 1.5 works
            if (PP[Ip].x_present < MM[0].Xbox[0]/shrinkage) PP[Ip].x_present = MM[0].Xbox[0]/shrinkage;
            if (PP[Ip].x_present > MM[0].Xbox[1]/shrinkage) PP[Ip].x_present = MM[0].Xbox[1]/shrinkage;
            if (PP[Ip].y_present < MM[0].Xbox[2]/shrinkage) PP[Ip].y_present = MM[0].Xbox[2]/shrinkage;
            if (PP[Ip].y_present > MM[0].Xbox[3]/shrinkage) PP[Ip].y_present = MM[0].Xbox[3]/shrinkage;
    */

    Ipx = Ip%maxGLnum;   // incase we are moving more particles than can be plotted
    Ipx = Ip%MAX_GLPARTICLES;
    pos[Ipx] = make_float4(scale*PP[Ip].x_present,-scale*PP[Ip].z_present,-scale*PP[Ip].y_present,  1.0f);

}  //other if iele>0 loop end    
}
else
{    PP[Ip].factor[0]=0.0; PP[Ip].factor[1]=0.0; PP[Ip].factor[2]=0.0;
    PP[Ip].i_ele =i_ele; 
    //return i_ele;   leave the particle loop
    igrounded++;
    //break;
}

   PP[Ip].time_now = time_now;
   // End of Particle loop on Ip
 
}

// End of a time step, increment to next  time_now += dt_sec;
// if time_frac >1, then it will fall out of the loop and not increment PP.timenow

time_now += dt_sec;
time_frac=(time_now - DD[DDT0].time)/(DD[DDT2].time - DD[DDT0].time);

}
//printf("end of move");
// end of move()
if ( cudaindex==0) DD[0].time_now = time_now;   // Only update dev_DD[] once
//  Hopefully the other cudas have gotten started by now and don't need to read dev_DD[0].time_now
}
//  End of move()


////////////////////////////////////////////////////////////////////////
///////////////////  move3d  ///////////////////////////////////////////
////////////////////////////////////////////////////////////////////////
//cuda kernel with four meshes and depth for 3d UVW's read from Field files
//uses MM[0:2] for the UVW and MM[2] to provide angle and depth
// move3d<<<  >>> ( pos,dev_P,dev_MM,dev_DD);

__global__
void move3d(float4 *pos, struct PPart *PP,struct MMesh *MM, struct DData *DD){
// Cuda Kernal to move the particles
// loop on all particles using cudaindex and stride
// for each particle find i_ele, depth angle     findiele
// interpolate sigma coordinate, find three corner values, average them to PP[iP].xyz
// Did that with all three time steps. Time interpolate
// Step PP[iP] position forward.

int IpTest=-250;
//int DeBuG = false;   //   true or false

/*   real stuff now  */
//float time_now;       //    now a passed argument
//float time_now = DD[0].time_now;    // Will use dev_DD after the first pass with new DD
//float time_frac=(time_now - DD[DDT0].time)/(DD[DDT2].time - DD[DDT0].time);
double dt_sec=DT_SEC;
//float deg2pi = 3.1415926/180.;

//  Cuda strides
int cudaindex = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

//if (cudaindex==-1 | cudaindex==-2){  //  0 2500
//    printf(" t.x=%d blockIdx.x=%d blockDim.x=%d gridDim.x=%d cudaindex=%d stride=%d\n",
//    threadIdx.x,blockIdx.x,blockDim.x, gridDim.x,cudaindex,stride);
//}

// Main time loop. Loop CUDA_STEPS times between returns for plotting
double  time_now = DD[0].time_now;
//if (cudaindex==0) printf("move3d %d, time_now= %fs %gh\n",cudaindex,time_now,time_now/3600.);
for (int itime=0; itime<CUDA_STEPS; itime++){
    
    
    for(int Ip = cudaindex; Ip <NUM_PARTICLES; Ip += stride){
        
        // Find surrounding triangle of Particle for all three meshes
        //  PP[Ip].i_ele4[iMM]
        //  PP[Ip].factor4[iMM][0:2]
        for (int iMM=0; iMM<3; iMM++) 
        {
          /*  if (Ip==-1 && itime==0) {
                printf("move3d before findiele itime= %d, time_now= %fs %gh\n",itime,time_now,time_now/3600.);
                for (int i_ele=0; i_ele<47184; i_ele+=12500)
                printf(" iMM = %d ele_func_tripart i_ele = %d ; triconnect = %ld\n",
                    iMM, i_ele, MM[iMM].tri_connect[i_ele][0]);
            }
          */
          //    findiele(Ip,iMM,PP,MM); 
                findielefake(Ip,iMM,PP,MM); 

        }  
        PP[Ip].answer[0]=0.0;

        if (Ip==IpTest && itime==0) printf(" move3d finished findielefake %d\n",itime);
        // interpolate values for angle and depth at PP[Ip].x,y    
        float VAR[3];
        int iMM=2;    //  mesh for w, angle and depth
        for (int i=0; i<3; i++) 
        { // i_ele is the element, ele[i_ele[0:2] are the nodes at corners of triangle i_ele
            long elei = MM[iMM].ele[PP[Ip].i_ele4[iMM]][i];
            VAR[i]=MM[iMM].ANGLE[elei];
        }
        if (Ip==IpTest && itime==0) printf("move3d before Interpolate2D findiele itime= %d, time_now= %fs %gh\n",itime,time_now,time_now/3600.);
        Interpolate2D(Ip,iMM,PP,VAR); 
        float angle=PP[Ip].answer[0];
        //or
        iMM=2; for (int i=0; i<3; i++) VAR[i]=MM[iMM].depth[MM[iMM].ele[PP[Ip].i_ele4[iMM]][i]];
        if (Ip==IpTest) printf(" depths = %g %g %g \n",VAR[0],VAR[1],VAR[2]);
        Interpolate2D(Ip,iMM,PP,VAR);  
        float depth=PP[Ip].answer[0]; 
        if (Ip==IpTest && itime==0) printf("move3d after Interpolate2D angle[%d]=%g  depth=%g\n",Ip,angle,depth);

        // Find the isigmap, isigmam and sigmafrac 
        //  U[iP] = U[isigmap]*sigmafrac +U[isigmam]*(1.0-sigmafrac)
        // do three times and use timefrac to produce final VAR[3] at corners
        // iMM = 0U  1V  2W   
        // Only works for UVW.  In future add special cases 3T 4S iMM=iMM-1 or -2 
        if (Ip==IpTest && itime==0) 
          printf("move3d before Interpolatesigma itime= %d, time_now= %fs %gh\n"
               ,itime,time_now,time_now/3600.);
        iMM=0; Interpolatesigma(Ip, iMM, PP, DD, MM, depth,time_now); 
        float Up=PP[Ip].answer[0];
        iMM=1; Interpolatesigma(Ip, iMM, PP, DD, MM, depth,time_now); 
        float Vp=PP[Ip].answer[0];
        //Up = 1.0; 
        //Vp = 0.0; 
        float cosa = cos(angle);
        float sina = sin(angle);
        float Upnow = cosa*Up -sina*Vp;
        float Vpnow = sina*Up +cosa*Vp;
        iMM=2; Interpolatesigma(Ip, iMM, PP, DD, MM, depth,time_now); 
        float Wpnow=PP[Ip].answer[0];
        //Wpnow = 0.0;
        if (Ip==IpTest && itime==0) printf("move3d after sigma UVp[%d]= %g %g UVWpnow= %g, %g, %g  angle=%g  depth=%g\n"
           ,Ip,Up,Vp,Upnow, Vpnow, Wpnow,angle,depth);

        
        /*  Now have time and space interpolates of U,V,W for particle */
        /* Apply them to the particle coordinates and done! 
        (unless temporal runge kutta is needed. 
            Running goofy small time steps)*/
            
            PP[Ip].x_present += dt_sec*(Upnow*1.) ; 
            PP[Ip].y_present += dt_sec*(Vpnow*1.); 
            PP[Ip].z_present += dt_sec*Wpnow*1.0;
            PP[Ip].z_present = min(PP[Ip].z_present, -0.01);       // if z_p is above -0.01
            PP[Ip].z_present = max(PP[Ip].z_present, -depth);    // if z_p is below -depth

            // End of Particle loop on Ip
        }
        
        // End of a time step, increment to next  time_now += dt_sec;
        // if time_frac >1, then it will fall out of the loop and not increment PP.timenow
        //if (cudaindex==2) printf(" 1=%lf",time_now);

        time_now+=dt_sec;    
        //if (cudaindex==2 && itime==0) printf("move3d end timeloop itime= %d, time_now= %lfs %lfh\n",itime,time_now,time_now/3600.);
        //if (cudaindex==2 && itime==(CUDA_STEPS-1)) printf("move3d end timeloop itime= %d, time_now= %lfs %lfh\n",itime,time_now,time_now/3600.);
        //if (cudaindex==2) printf(" 2=%lf",time_now);
    }
    // Update the VBO  pos[]
    //unsigned int maxGLnum = blockDim.x*blockDim.y*gridDim.x*gridDim.y;
    //maxGLnum = MAX_GLPARTICLES;
    float scale = SCALE_GL;
    for(int Ip = cudaindex; Ip <NUM_PARTICLES; Ip += stride){
        int Ipx = Ip%MAX_GLPARTICLES;
        pos[Ipx] = make_float4(scale*PP[Ip].x_present,1000.*scale*PP[Ip].z_present,-scale*PP[Ip].y_present,  1.0f);
    }
    
    if (cudaindex==-1){
        printf("end of move,  pos[0][0]=%g\n",pos[cudaindex]);
    }
    // end of move()
    if ( cudaindex==0) DD[0].time_now = time_now;   // Only update dev_DD[] once
    //  Hopefully the other cudas have gotten started by now and don't need to read dev_DD[0].time_now
}

/////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////
// update which triangle the iP is in for the three meshes   iMM 012 UVW  
//           note the iMM=2 also gives iele and factors for ANGLE and depth
// Sets PP[Ip].i_ele4[iMM] and the factors  PP[Ip].factor4[iMM][0:2] 
// for (in iMM=0; iMM<3; iMM++)
__device__ void findielefake(int Ip,int iMM,struct PPart *PP, struct MMesh *MM)
{


    int i_ele, keepgoing, k;
    float xpart, ypart;
    float smallest_value = -0.01000; //  -.01  -0.001; 

    // Find surrounding triangle of Particle
    i_ele = PP[Ip].i_ele4[iMM];
    xpart = PP[Ip].x_present;
    ypart = PP[Ip].y_present;
    //return;
    //if(Ip==0) printf(" start findiele i_ele=%d \n",i_ele);
    //  Check for out of domain/ grounded particle
    //  do work if in-domain  else increment igrounded and skip main part of move
    if (i_ele >= 0) { 
        
        keepgoing = 1; 
        while (keepgoing  > 0 ){
            
            //if(Ip==0) printf(" keepgoing findielefake i_ele=%d ",i_ele);

            //  if any of the f's are negative, walk that way and restart while loop
            k=0;
            PP[Ip].factor4[iMM][k]=MM[iMM].a_frac[i_ele][k]*xpart + 
            MM[iMM].b_frac[i_ele][k]*ypart + MM[iMM].c_frac[i_ele][k];
            if ( PP[Ip].factor4[iMM][k] < smallest_value) { 
                i_ele = MM[iMM].tri_connect[i_ele][0]; 
            }
            else { 
                k=1;
                PP[Ip].factor4[iMM][k]=MM[iMM].a_frac[i_ele][k]*xpart + MM[iMM].b_frac[i_ele][k]*ypart + MM[iMM].c_frac[i_ele][k];
                if ( PP[Ip].factor4[iMM][k] < smallest_value ) { 
                  i_ele = MM[iMM].tri_connect[i_ele][1] ; 
            }
            else { 
                k=2;
                PP[Ip].factor4[iMM][k]=MM[iMM].a_frac[i_ele][k]*xpart + MM[iMM].b_frac[i_ele][k]*ypart + MM[iMM].c_frac[i_ele][k];
                if ( PP[Ip].factor4[iMM][k] < smallest_value ) { 
                i_ele = MM[iMM].tri_connect[i_ele][2] ;
                }
                else {
                   //  Found it, iele,   all f's are positive 
                   keepgoing = 0;
            }
            }
         }
         if (i_ele < 0) {    // newly grounded particle, zero him out.
               PP[Ip].factor4[iMM][0]=0.0; PP[Ip].factor4[iMM][1]=0.0; PP[Ip].factor4[iMM][2]=0.0;
               PP[Ip].i_ele4[iMM] = i_ele;
               keepgoing = 0;
         }
         if (keepgoing>0) keepgoing++;
         if (keepgoing > 500) { //printf(" k%d  ",Ip);
         i_ele=-1;
         PP[Ip].i_ele4[iMM] = -1;
         PP[Ip].x_present=0.0;
         PP[Ip].y_present=0.0;
         PP[Ip].z_present=0.0;

         keepgoing=0;}
       }   
       
       //return;
       if (i_ele>=0){     // good particle still in the mesh
        PP[Ip].i_ele4[iMM]=i_ele;}
       // end of while keepgoing 
      

}      
//if(Ip==0) printf(" end findielefake i_ele=%d \n",i_ele);
 




    return;
}
__device__ void findiele(int Ip,int iMM,struct PPart *PP, struct MMesh *MM)
{
    int i_ele, keepgoing, k;
    float xpart, ypart;
    float smallest_value = -0.01000; // -0.001; 

    // Find surrounding triangle of Particle
    i_ele = PP[Ip].i_ele4[iMM];
    xpart = PP[Ip].x_present;
    ypart = PP[Ip].y_present;
    //if(Ip==0) printf(" start findiele i_ele=%d \n",i_ele);
    //  Check for out of domain/ grounded particle
    //  do work if in-domain  else increment igrounded and skip main part of move
    if (i_ele >= 0) { 
        
        keepgoing = 1; 
        while (keepgoing  == 1){
            
            //if(Ip==0) printf(" keepgoing findiele i_ele=%d \n",i_ele);

            //  if any of the f's are negative, walk that way and restart while loop
            k=0;
            PP[Ip].factor4[iMM][k]=MM[iMM].a_frac[i_ele][k]*xpart + 
            MM[iMM].b_frac[i_ele][k]*ypart + MM[iMM].c_frac[i_ele][k];
            if ( PP[Ip].factor4[iMM][k] < smallest_value) { 
                i_ele = MM[iMM].tri_connect[i_ele][0]; 
            }
            else { 
                k=1;
                PP[Ip].factor4[iMM][k]=MM[iMM].a_frac[i_ele][k]*xpart + MM[iMM].b_frac[i_ele][k]*ypart + MM[iMM].c_frac[i_ele][k];
                if ( PP[Ip].factor4[iMM][k] < smallest_value ) { 
                  i_ele = MM[iMM].tri_connect[i_ele][1] ; 
            }
            else { 
                k=2;
                PP[Ip].factor4[iMM][k]=MM[iMM].a_frac[i_ele][k]*xpart + MM[iMM].b_frac[i_ele][k]*ypart + MM[iMM].c_frac[i_ele][k];
                if ( PP[Ip].factor4[iMM][k] < smallest_value ) { 
                i_ele = MM[iMM].tri_connect[i_ele][2] ;
                }
                else {
                   //  Found it, iele,   all f's are positive 
                   keepgoing = 0;
            }
            }
         }
         if (i_ele < 0) {    // newly grounded particle, zero him out.
               PP[Ip].factor4[iMM][0]=0.0; PP[Ip].factor4[iMM][1]=0.0; PP[Ip].factor4[iMM][2]=0.0;
               PP[Ip].i_ele = i_ele;
               keepgoing = 0;
         }
       }   // end of while keepgoing 
       
       PP[Ip].i_ele4[iMM]=i_ele;

}      
//if(Ip==0) printf(" end findiele i_ele=%d \n",i_ele);

}


/////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////
// find 2d interpolated ANGLE(icase==0), depth(icase==1)
//  input is X,Y, A of i_ele points along with factor4
// MM[iMM].X[i_ele4[0:2]] MM[iMM].Y[i_ele4[0:2]] MM[iMM].ANGLE[i_ele4[0:2]] 
// PP[Ip].factor4[iMM][i_ele[0:]]    
// 2Dinterpolate(Ip,iMM,PP,MM,icase);  // icase = 0U, 1V, 2W, 3ANGLE, 4depth  
//    maybe do  VAR[3] = MM[iMM].ANGLE[i_ele4[0:2]]  instead of icase 
//    That way we can feed it the vertical interpolates of UVW[3]
//float VAR[3];
//iMM=3; for (int i=0; i<3; i++) VAR[i]=MM[iMM].angle[PP[Ip].iele4[iMM][i]];
//float angle = 2Dinterpolate(Ip,iMM,PP,MM,VAR);
//iMM=4; for (int i=0; i<3; i++) VAR[i]=MM[iMM].depth[PP[Ip].iele4[iMM][i]];
//float depth = 2Dinterpolate(Ip,iMM,PP,MM,VAR); 

__device__ void Interpolate2D(int Ip, int iMM, struct PPart *PP, float *VAR)
{
    /*
    iMM=3; 
    for (int i=0; i<3; i++) 
    { // i_ele is the element, ele[i_ele[0:2] are the nodes at corners of triangle i_ele
        long elei = MM[iMM].ele[PP[Ip].iele4[iMM][i]];
        VAR[i]=MM[iMM].angle[elei];
    }
    float angle = 2Dinterpolate(Ip,iMM,PP,MM,VAR);
    */

    float factor0=PP[Ip].factor4[iMM][0];
    float factor1=PP[Ip].factor4[iMM][1];
    float factor2=PP[Ip].factor4[iMM][2];
    
   PP[Ip].answer[0] = factor0*VAR[0]+factor1*VAR[1]+factor2*VAR[2];

}


/////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////

__device__ void Interpolatesigma(int Ip, int iMM, 
    struct PPart *PP, struct DData *DD, struct MMesh *MM, float depth, float time_now )
{
    // Find the isigmap, isigmam and sigmafrac 
    //  U[iP] = U[isigmap]*sigmafrac +U[isigmam]*(1.0-sigmafrac)
    // do three times and use timefrac to produce final VAR[3] at corners
    // iMM = 0U  1V  2W   
    // Only works for UVW.  In future add special cases 3T 4S iMM=iMM-1 or -2     int DDT0, DDT2;
    int IpTest = -250;

    int i_ele = PP[Ip].i_ele4[iMM];   
    float vart[3];
    float var[3]; 
    int sp, sm;

    float sigIp = PP[Ip].z_present / depth ;    // 0 to -1.0
    //  count up in sp to walk down in depth
    sp=1;
    while(MM[iMM].sigma[sp]< sigIp) sp++;  // increment if sp is still above sigIp
    sm = sp-1;                         // sp is below sigIp,  sm is above
    float sigfrac = (sigIp-MM[iMM].sigma[sp])/(MM[iMM].sigma[sm]- MM[iMM].sigma[sp]);
    
    // Pick out the three DD's to interpolate in time
    int DD3[3];
    DD3[0]=DD[0].DD3[0];
    DD3[1]=DD[0].DD3[1];
    DD3[2]=DD[0].DD3[2];
    int DDT0=DD3[0];
    //DDT1=DD3[1];
    int DDT2=DD3[2];
    
    if (Ip==IpTest  ) printf(" start of interpretsigma iMM=%d  z_present= %g /depth=%g =sigIP = %g \n  sm,sp sigma[%d]=%g sigma[%d]=%g sigIP %g sigfrac %g\n"
    ,iMM,PP[Ip].z_present, depth,sigIp,sm,MM[iMM].sigma[sm],sp,MM[iMM].sigma[sp],sigIp,sigfrac);
    // loop on time DD3[i]
    // loop on three corners ele[i_ele][j]
    // average sm and sp at the corner
    
    for (int it=0; it<3; it++){  // time loop for DD3[it]
        for (int j=0; j<3; j++){     // loop around corners to get sigma averaged variable
            long ele0=MM[iMM].ele[i_ele][j];
            if      (iMM==0){ // U
                var[j] = DD[DD3[it]].U[sm][ele0]*sigfrac  + DD[DD3[it]].U[sp][ele0]* (1.0 - sigfrac);
            }
            else if (iMM==1){ // V    
                var[j] = DD[DD3[it]].V[sm][ele0]*sigfrac  + DD[DD3[it]].V[sp][ele0]* (1.0 - sigfrac);
            }
            else if (iMM==2){ // W    
                var[j] = DD[DD3[it]].W[sm][ele0]*sigfrac  + DD[DD3[it]].W[sp][ele0]* (1.0 - sigfrac);
            }
            else { printf(" \n\n Bad iMM in Interpolatesigma %d\n\n",iMM); }
        }
        // Have sigma average var[0:2] at the three corners
        //interpolate to center, to get three time increments vart[0:2] 
        //float factor0=PP[Ip].factor4[iMM][0];
        //float factor1=PP[Ip].factor4[iMM][1];
        //float factor2=PP[Ip].factor4[iMM][2];
        vart[it]= PP[Ip].factor4[iMM][0]*var[0] 
        + PP[Ip].factor4[iMM][1]*var[1]
        + PP[Ip].factor4[iMM][2]*var[2];
        
        if (Ip==IpTest ) printf("  intersig DD3=%d var=%g %g %g vart=%g\n "
             ,DD3[it],var[0],var[1],var[2],vart[it]);
    }
    // Finally interpolate in time to get final answer for U, V, W to mover PP[Ip]
    // float time_now = DD[0].time_now;    // Will use dev_DD after the first pass with new DD
    float time_frac=(time_now - DD[DDT0].time)/(DD[DDT2].time - DD[DDT0].time);
    
    //float a =  2.*vart[2] -4.*vart[1] +2.*vart[0];
    //float b = -   vart[2] +4.*vart[1] +   vart[0];
    //float c =                       vart[0];
    //float Upnow = a*time_frac*time_frac + b*time_frac + c;
float Upnow = ( 2.*vart[2] -4.*vart[1] +2.*vart[0])*time_frac*time_frac 
             +(-   vart[2] +4.*vart[1] +   vart[0])*time_frac 
             +(                      vart[0]);                



/*  Now have time sigma and space interpolates of U,V,W for particle */
PP[Ip].answer[0] = Upnow;
        if (Ip==IpTest ) printf("  intersigend timenow=%fs timefrac=%g Upnow=%g\n ",time_now,time_frac,Upnow);


}